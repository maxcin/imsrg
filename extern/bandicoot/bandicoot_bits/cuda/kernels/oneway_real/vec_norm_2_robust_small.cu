#include "hip/hip_runtime.h"
// Copyright 2023 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

__global__
void
COOT_FN(PREFIX,vec_norm_2_robust_small)(const eT1* in_mem,
                                        const UWORD n_elem,
                                        eT1* out_mem)
  {
  // This kernel is not used!  It is just a placeholder.
  // Only the OpenCL version is used.
  }
