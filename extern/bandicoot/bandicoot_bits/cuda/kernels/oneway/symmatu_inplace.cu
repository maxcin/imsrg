#include "hip/hip_runtime.h"
// Copyright 2023 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------



__global__
void
COOT_FN(PREFIX,symmatu_inplace)(eT1* out,
                                const UWORD size) // matrix is expected to be square
  {
  const UWORD row = blockIdx.x * blockDim.x + threadIdx.x;
  const UWORD col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < size && col < size && col > row)
    {
    const eT1 val = out[row + size * col];

    // only need to copy to the lower triangle for the in-place version
    out[col + size * row] = val;
    }
  }
