#include "hip/hip_runtime.h"
// Copyright 2019 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

__global__
void
COOT_FN(PREFIX,fill)(eT1* out,
                     const eT1 val,
                     const UWORD n_rows,
                     const UWORD n_cols,
                     const UWORD M_n_rows)
  {
  const UWORD row = blockIdx.x * blockDim.x + threadIdx.x;
  const UWORD col = blockIdx.y * blockDim.y + threadIdx.y;
  const UWORD index = col * M_n_rows + row;

  if(row < n_rows && col < n_cols)
    {
    out[index] = val;
    }
  }
