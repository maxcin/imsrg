#include "hip/hip_runtime.h"
// Copyright 2019 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

__global__
void
COOT_FN(PREFIX,inplace_set_eye)(eT1* out,
                                const UWORD n_rows,
                                const UWORD n_cols)
  {
  const UWORD row = blockIdx.x * blockDim.x + threadIdx.x;
  const UWORD col = blockIdx.y * blockDim.y + threadIdx.y;
  if( (row < n_rows) && (col < n_cols) )
    {
    const UWORD offset = row + col * n_rows;
    out[offset] = (row == col) ? (eT1)(1) : (eT1)(0);
    }
  }
