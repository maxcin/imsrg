#include "hip/hip_runtime.h"
// Copyright 2023 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

// this kernel is technically incorrect if the size is not a factor of 2!
__global__
void
COOT_FN(PREFIX,approx_equal_small)(uint* out_mem,
                                   const eT1* A_mem,
                                   const UWORD A_M_n_rows,
                                   const eT1* B_mem,
                                   const UWORD B_M_n_rows,
                                   const UWORD n_rows,
                                   const UWORD n_elem,
                                   const UWORD mode,
                                   const eT1 abs_tol,
                                   const eT1 rel_tol)
  {
  uint* aux_mem = (uint*) aux_shared_mem;

  const UWORD tid = threadIdx.x;
  UWORD i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  const UWORD grid_size = blockDim.x * 2 * gridDim.x;

  aux_mem[tid] = 1;

  while (i + blockDim.x < n_elem)
    {
    // A bit painful...
    const UWORD row1 = i % n_rows;
    const UWORD col1 = i / n_rows;
    const UWORD row2 = (i + blockDim.x) % n_rows;
    const UWORD col2 = (i + blockDim.x) / n_rows;

    const UWORD A_loc1 = row1 + col1 * A_M_n_rows;
    const UWORD A_loc2 = row2 + col2 * A_M_n_rows;
    const UWORD B_loc1 = row1 + col1 * B_M_n_rows;
    const UWORD B_loc2 = row2 + col2 * B_M_n_rows;

    const eT1 A_val1 = A_mem[A_loc1];
    const eT1 B_val1 = B_mem[B_loc1];
    const eT1 A_val2 = A_mem[A_loc2];
    const eT1 B_val2 = B_mem[B_loc2];

    if (coot_isnan(A_val1) || coot_isnan(B_val1) || coot_isnan(A_val2) || coot_isnan(B_val2))
      {
      // Not approximately equal.
      aux_mem[tid] &= 0;
      }

    const eT1 absdiff1 = COOT_FN(PREFIX,absdiff)(A_val1, B_val1);
    const eT1 absdiff2 = COOT_FN(PREFIX,absdiff)(A_val2, B_val2);

    if ((mode & 1) == 1) // absolute
      {
      aux_mem[tid] &= (absdiff1 <= abs_tol);
      aux_mem[tid] &= (absdiff2 <= abs_tol);
      }

    if ((mode & 2) == 2) // relative
      {
      const eT1 max_val1 = max(ET1_ABS(A_val1), ET1_ABS(B_val1));
      const eT1 max_val2 = max(ET1_ABS(A_val2), ET1_ABS(B_val2));

      if (max_val1 >= (eT1) 1)
        {
        aux_mem[tid] &= (absdiff1 <= rel_tol * max_val1);
        aux_mem[tid] &= (absdiff2 <= rel_tol * max_val2);
        }
      else
        {
        aux_mem[tid] &= (absdiff1 / max_val1 <= rel_tol);
        aux_mem[tid] &= (absdiff2 / max_val2 <= rel_tol);
        }
      }

    i += grid_size;
    }
  if (i < n_elem)
    {
    const UWORD row = i % n_rows;
    const UWORD col = i / n_rows;

    const UWORD A_loc = row + col * A_M_n_rows;
    const UWORD B_loc = row + col * B_M_n_rows;

    const eT1 A_val = A_mem[A_loc];
    const eT1 B_val = B_mem[B_loc];

    if (coot_isnan(A_val) || coot_isnan(B_val))
      {
      // Not approximately equal.
      aux_mem[tid] &= 0;
      }

    const eT1 absdiff = COOT_FN(PREFIX,absdiff)(A_val, B_val);

    if ((mode & 1) == 1) // absolute
      {
      aux_mem[tid] &= (absdiff <= abs_tol);
      }

    if ((mode & 2) == 2) // relative
      {
      const eT1 max_val = max(ET1_ABS(A_val), ET1_ABS(B_val));

      if (max_val >= (eT1) 1)
        {
        aux_mem[tid] &= (absdiff <= rel_tol * max_val);
        }
      else
        {
        aux_mem[tid] &= (absdiff / max_val <= rel_tol);
        }
      }
    }

  for (UWORD s = blockDim.x / 2; s > 0; s >>= 1)
    {
    if (tid < s)
      {
      aux_mem[tid] &= aux_mem[tid + s];
      }
  }

  if (tid == 0)
    {
    out_mem[blockIdx.x] = aux_mem[0];
    }
  }
