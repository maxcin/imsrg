#include "hip/hip_runtime.h"
// Copyright 2023 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

__global__
void
COOT_FN(PREFIX,rel_any_neq_rowwise)(UWORD* out,
                                    const eT1* A,
                                    const eT2 val,
                                    const UWORD A_n_rows,
                                    const UWORD A_n_cols)
  {
  const UWORD row = blockIdx.x * blockDim.x + threadIdx.x;
  if(row < A_n_rows)
    {
    UWORD result = 0;
    for (UWORD i = 0; i < A_n_cols; ++i)
      {
      const eT2 val1 = (eT2) A[i * A_n_rows + row];
      result |= (val1 != val);
      if (result == 1)
        break;
      }

    out[row] = result;
    }
  }
