#include "hip/hip_runtime.h"
// Copyright 2023 Ryan Curtin (http://www.ratml.org/)
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// ------------------------------------------------------------------------

__global__
void
COOT_FN(PREFIX,min_colwise_conv_post)(eT2* dest,
                                      const eT1* src,
                                      const UWORD n_rows,
                                      const UWORD n_cols,
                                      const UWORD dest_mem_incr,
                                      const UWORD src_M_n_rows)
  {
  const UWORD col = blockIdx.x * blockDim.x + threadIdx.x;
  if(col < n_cols)
    {
    const eT1* colptr = &(src[col * src_M_n_rows]);
    eT1 acc = (eT1) colptr[0];
    for (UWORD i = 1; i < n_rows; ++i)
      {
      acc = min(acc, colptr[i]);
      }

    dest[col * dest_mem_incr] = (eT2) (acc);
    }
  }
